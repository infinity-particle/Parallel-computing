#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <intrin.h>
#include <ctime>

using namespace std;

#define MATRIX_SIZE 1024
#define BlockSize 32

void matrixMultiplicationWithCuda(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE]);
void matrixMultiplicationCPU(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE]);
void checkCUDAStatus(hipError_t cudaStatus);
bool compareResults(int cudaMultiplicationResult[][MATRIX_SIZE],int cpuMultiplicationResult[][MATRIX_SIZE]);

__global__ void matrixMultiplicationKernel(int *A, int *B, int *C)
{
	int result = 0;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;


	if (row > MATRIX_SIZE || column > MATRIX_SIZE) return;

	for (int i = 0; i < MATRIX_SIZE; i++){
		result += A[row*MATRIX_SIZE + i] * B[i*MATRIX_SIZE + column];
	}
	C[row*MATRIX_SIZE + column] = result;
}

int main()
{
    srand(time(0));
	auto matrixA = new int[MATRIX_SIZE][MATRIX_SIZE];
	auto matrixB = new int[MATRIX_SIZE][MATRIX_SIZE];
	auto cudaMultiplicationResult = new int[MATRIX_SIZE][MATRIX_SIZE];
	auto cpuMultiplicationResult = new int[MATRIX_SIZE][MATRIX_SIZE];

	for (int i = 0; i<MATRIX_SIZE; i++){
		for (int j = 0; j < MATRIX_SIZE; j++){
			matrixA[i][j] = rand() % 2048;
			matrixB[i][j] = rand() % 2048;
			cpuMultiplicationResult[i][j] = 0;
		}
	}
	

	matrixMultiplicationWithCuda(matrixA, matrixB, cudaMultiplicationResult);
	matrixMultiplicationCPU(matrixA, matrixB, cpuMultiplicationResult);
	if(compareResults(cudaMultiplicationResult, cpuMultiplicationResult)){
		printf("Results are equals!\n");
	}else{
		printf("Results are NOT equals!\n");
	}
	system("pause");
	delete[] matrixA;
	delete[] matrixB;
	delete[] cudaMultiplicationResult;
	delete[] cpuMultiplicationResult;
}

void matrixMultiplicationWithCuda(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE])
{
    int *dev_a, *dev_b, *dev_c;
	clock_t begin, end;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_a, ((MATRIX_SIZE)*(MATRIX_SIZE))*sizeof(int));
	checkCUDAStatus(cudaStatus);
	cudaStatus = hipMalloc((void**)&dev_b, ((MATRIX_SIZE)*(MATRIX_SIZE))*sizeof(int));
	checkCUDAStatus(cudaStatus);
	cudaStatus = hipMalloc((void**)&dev_c, ((MATRIX_SIZE)*(MATRIX_SIZE))*sizeof(int));



	cudaStatus = hipMemcpy(dev_a, A, ((MATRIX_SIZE*MATRIX_SIZE))*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAStatus(cudaStatus);
	cudaStatus = hipMemcpy(dev_b, B, ((MATRIX_SIZE*MATRIX_SIZE))*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAStatus(cudaStatus);

	dim3 dimBlock(BlockSize, BlockSize);
	dim3 dimGrid((MATRIX_SIZE + dimBlock.x - 1) / dimBlock.x, (MATRIX_SIZE + dimBlock.y - 1) / dimBlock.y);
	
	begin = clock();
	matrixMultiplicationKernel <<< dimGrid, dimBlock >>>(dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	end = clock();

	cudaStatus = hipGetLastError();
	checkCUDAStatus(cudaStatus);

	cudaStatus = hipMemcpy(C, dev_c, ((MATRIX_SIZE*MATRIX_SIZE))*sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAStatus(cudaStatus);
	printf("CUDA time: %lf seconds\n", (double)(end - begin)/CLOCKS_PER_SEC);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

void matrixMultiplicationCPU(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE]){
	clock_t begin, end;
	begin = clock();
	for (int row = 0; row < MATRIX_SIZE; row++) {  
		for (int col = 0; col < MATRIX_SIZE; col++) {  
			for (int inner = 0; inner < MATRIX_SIZE; inner++) {  
				C[row][col] += A[row][inner] * B[inner][col];  
			}   
		}  
	} 
	end = clock();
	printf("CPU time: %lf seconds\n", (double)(end - begin)/CLOCKS_PER_SEC);
}

void checkCUDAStatus(hipError_t cudaStatus){
	if(cudaStatus != hipSuccess){
		printf("CUDA return error code: %d", cudaStatus);
		exit(-1);
	}
}

bool compareResults(int cudaMultiplicationResult[][MATRIX_SIZE],int cpuMultiplicationResult[][MATRIX_SIZE]){
	for(int row = 0; row < MATRIX_SIZE; row++){
		for(int column = 0; column < MATRIX_SIZE; column++){
			if(cudaMultiplicationResult[row][column] != cpuMultiplicationResult[row][column]){
				return false;
			}
		}
	}
	return true;
}