#include "hip/hip_runtime.h"
#include <iostream> 
#include <hip/hip_runtime.h> 
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <intrin.h>
#include <ctime>
 
#pragma comment(lib, "cudart") 
 
using namespace std;

#define MATRIX_SIZE 1024
#define BlockSize 32
 
// CUDA kernel: cubes each array value 
void matrixMultiplicationWithCuda(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE], bool flagOptimozation);
void matrixMultiplicationCPU(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE]);
void checkCUDAStatus(hipError_t cudaStatus);
bool compareResults(int cudaMultiplicationResult[][MATRIX_SIZE],int cpuMultiplicationResult[][MATRIX_SIZE]);

__global__ void matrixMultiplicationKernel(int *A, int *B, int *C)
{
	int result = 0;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	//__syncthreads();
	if (row > MATRIX_SIZE || column > MATRIX_SIZE) return;

	for (int i = 0; i < MATRIX_SIZE; i++){
		result += A[row*MATRIX_SIZE + i] * B[i*MATRIX_SIZE + column];
	}
	C[row*MATRIX_SIZE + column] = result;
}

__global__ void matrixMultiplicationWithOptimizationKernel(int *A, int *B, int *C)
{
	
	__shared__ float ds_M[BlockSize][BlockSize];
    __shared__ float ds_N[BlockSize][BlockSize];
    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * BlockSize + ty,
       Col = bx * BlockSize + tx;
    float Pvalue = 0;

    for (int m = 0; m < (MATRIX_SIZE-1)/BlockSize+1; ++m) {
       if (Row < MATRIX_SIZE && m*BlockSize+tx < MATRIX_SIZE)
          ds_M[ty][tx] = A[Row*MATRIX_SIZE + m*BlockSize+tx];
       else
          ds_M[ty][tx] = 0;
       if (Col < MATRIX_SIZE && m*BlockSize+ty < MATRIX_SIZE)
          ds_N[ty][tx] = B[(m*BlockSize+ty)*MATRIX_SIZE+Col];
       else
          ds_N[ty][tx] = 0;

       __syncthreads();
       for (int k = 0; k < BlockSize; ++k)
          Pvalue += ds_M[ty][k] * ds_N[k][tx];
       __syncthreads();
    }
    if (Row < MATRIX_SIZE && Col < MATRIX_SIZE)
       C[Row*MATRIX_SIZE+Col] = Pvalue;
}
		
int main()
{
    srand(time(0));
	auto matrixA = new int[MATRIX_SIZE][MATRIX_SIZE];
	auto matrixB = new int[MATRIX_SIZE][MATRIX_SIZE];
	auto cudaMultiplicationResult = new int[MATRIX_SIZE][MATRIX_SIZE];
	auto cudaWithOptimizationMultiplicationResult = new int[MATRIX_SIZE][MATRIX_SIZE];
	auto cpuMultiplicationResult = new int[MATRIX_SIZE][MATRIX_SIZE];

	for (int i = 0; i<MATRIX_SIZE; i++){
		for (int j = 0; j < MATRIX_SIZE; j++){
			matrixA[i][j] = rand() % 2048;
			matrixB[i][j] = rand() % 2048;
			cpuMultiplicationResult[i][j] = 0;
		}
	}
	
	matrixMultiplicationWithCuda(matrixA, matrixB, cudaMultiplicationResult, false);
	matrixMultiplicationWithCuda(matrixA, matrixB, cudaWithOptimizationMultiplicationResult, true);
	matrixMultiplicationCPU(matrixA, matrixB, cpuMultiplicationResult);
	
	if(compareResults(cudaMultiplicationResult, cpuMultiplicationResult)){
		printf("Results are equals!\n");
	}else{
		printf("Results are NOT equals!\n");
	}
	system("pause");
	delete[] matrixA;
	delete[] matrixB;
	delete[] cudaMultiplicationResult;
	delete[] cpuMultiplicationResult;
}



void matrixMultiplicationWithCuda(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE], bool flagOptimization)
{
    int *dev_a, *dev_b, *dev_c;
	clock_t begin, end;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&dev_a, ((MATRIX_SIZE)*(MATRIX_SIZE))*sizeof(int));
	checkCUDAStatus(cudaStatus);
	cudaStatus = hipMalloc((void**)&dev_b, ((MATRIX_SIZE)*(MATRIX_SIZE))*sizeof(int));
	checkCUDAStatus(cudaStatus);
	cudaStatus = hipMalloc((void**)&dev_c, ((MATRIX_SIZE)*(MATRIX_SIZE))*sizeof(int));



	cudaStatus = hipMemcpy(dev_a, A, ((MATRIX_SIZE*MATRIX_SIZE))*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAStatus(cudaStatus);
	cudaStatus = hipMemcpy(dev_b, B, ((MATRIX_SIZE*MATRIX_SIZE))*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAStatus(cudaStatus);

	dim3 dimBlock(BlockSize, BlockSize);
	dim3 dimGrid((MATRIX_SIZE + dimBlock.x - 1) / dimBlock.x, (MATRIX_SIZE + dimBlock.y - 1) / dimBlock.y);
	
	begin = clock();
	if(flagOptimization)
		matrixMultiplicationWithOptimizationKernel <<< dimGrid, dimBlock >>>(dev_a, dev_b, dev_c);
	else
		matrixMultiplicationKernel <<< dimGrid, dimBlock >>>(dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	end = clock();

	cudaStatus = hipGetLastError();
	checkCUDAStatus(cudaStatus);

	cudaStatus = hipMemcpy(C, dev_c, ((MATRIX_SIZE*MATRIX_SIZE))*sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAStatus(cudaStatus);

	if(flagOptimization)
		printf("CUDA time with optimization: %lf seconds\n", (double)(end - begin)/CLOCKS_PER_SEC);
	else
		printf("CUDA time: %lf seconds\n", (double)(end - begin)/CLOCKS_PER_SEC);	

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

void matrixMultiplicationCPU(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE]){
	clock_t begin, end;
	begin = clock();
	for (int row = 0; row < MATRIX_SIZE; row++) {  
		for (int col = 0; col < MATRIX_SIZE; col++) {  
			for (int inner = 0; inner < MATRIX_SIZE; inner++) {  
				C[row][col] += A[row][inner] * B[inner][col];  
			}   
		}  
	} 
	end = clock();
	printf("CPU time: %lf seconds\n", (double)(end - begin)/CLOCKS_PER_SEC);
}

void checkCUDAStatus(hipError_t cudaStatus){
	if(cudaStatus != hipSuccess){
		printf("CUDA return error code: %d", cudaStatus);
		exit(-1);
	}
}

bool compareResults(int cudaMultiplicationResult[][MATRIX_SIZE],int cpuMultiplicationResult[][MATRIX_SIZE]){
	for(int row = 0; row < MATRIX_SIZE; row++){
		for(int column = 0; column < MATRIX_SIZE; column++){
			if(cudaMultiplicationResult[row][column] != cpuMultiplicationResult[row][column]){
				return false;
			}
		}
	}
	return true;
}