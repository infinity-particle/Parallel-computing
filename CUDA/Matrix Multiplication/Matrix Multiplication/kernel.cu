#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <intrin.h>
#include <ctime>

using namespace std;

#define MATRIX_SIZE 1024
#define BlockSize 32

void matrixMultiplicationWithCuda(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE]);
void matrixMultiplicationCPU(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE]);

__global__ void matrixMultiplicationKernel(int *A, int *B, int *C)
{
	int result = 0;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;


	if (row > MATRIX_SIZE || col > MATRIX_SIZE) return;

	for (int i = 0; i < MATRIX_SIZE; i++){
		result += A[row*MATRIX_SIZE + i] * B[i*MATRIX_SIZE + col];
	}
	C[row*MATRIX_SIZE + col] = result;
}

int main()
{
    srand(time(0));
	auto A = new int[MATRIX_SIZE][MATRIX_SIZE];
	auto B = new int[MATRIX_SIZE][MATRIX_SIZE];
	auto C = new int[MATRIX_SIZE][MATRIX_SIZE];
	auto cpuMul = new int[MATRIX_SIZE][MATRIX_SIZE];

	for (int i = 0; i<MATRIX_SIZE; i++){
		for (int j = 0; j < MATRIX_SIZE; j++){
			A[i][j] = rand() % 100;
			B[i][j] = rand() % 100;
			cpuMul[i][j] = 0;
		}
	}
	

	matrixMultiplicationWithCuda(A,B,C);
	matrixMultiplicationCPU(A,B,cpuMul);
	system("pause");
	delete[]A;
	delete[]B;
	delete[]C;
}

// Helper function for using CUDA to add vectors in parallel.
void matrixMultiplicationWithCuda(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE])
{
    int *dev_a, *dev_b, *dev_c;
	unsigned long long begin, end;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//allocate memory on global memory of gpu
	hipError_t err = hipMalloc((void**)&dev_a, ((MATRIX_SIZE)*(MATRIX_SIZE))*sizeof(int));
	printf("Cuda malloc A:%s \n", hipGetErrorString(err));
	err = hipMalloc((void**)&dev_b, ((MATRIX_SIZE)*(MATRIX_SIZE))*sizeof(int));
	printf("Cuda malloc B:%s \n", hipGetErrorString(err));
	err = hipMalloc((void**)&dev_c, ((MATRIX_SIZE)*(MATRIX_SIZE))*sizeof(int));
	printf("Cuda malloc C:%s \n", hipGetErrorString(err));


	//Copy array A and B on device allocated memory
	err = hipMemcpy(dev_a, A, ((MATRIX_SIZE*MATRIX_SIZE))*sizeof(int), hipMemcpyHostToDevice);
	printf("Cuda memcpy to device A:%s \n", hipGetErrorString(err));
	err = hipMemcpy(dev_b, B, ((MATRIX_SIZE*MATRIX_SIZE))*sizeof(int), hipMemcpyHostToDevice);
	printf("Cuda memcpy to device B:%s \n", hipGetErrorString(err));

	//two dimension threads
	dim3 dimBlock(BlockSize, BlockSize);
	dim3 dimGrid((MATRIX_SIZE + dimBlock.x - 1) / dimBlock.x, (MATRIX_SIZE + dimBlock.y - 1) / dimBlock.y);
	
	//call the kernel function multi
	hipEventRecord(start);
	matrixMultiplicationKernel <<< dimGrid, dimBlock >> >(dev_a, dev_b, dev_c);
	hipEventRecord(stop);

	//retrieve array C from device memory
	err = hipMemcpy(C, dev_c, ((MATRIX_SIZE*MATRIX_SIZE))*sizeof(int), hipMemcpyDeviceToHost);
	printf("Cuda memcpy to HOST C:%s \n", hipGetErrorString(err));
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("CUDA time: %f ms\n", milliseconds);

	/*for (int i = 0; i < MATRIX_SIZE; i++){
		for (int j = 0; j < MATRIX_SIZE; j++){
			printf("C(%d,%d) = %d \n", i, j, C[i][j]);
		}
	}*/

	//free the memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

void matrixMultiplicationCPU(int A[][MATRIX_SIZE],int B[][MATRIX_SIZE],int C[][MATRIX_SIZE]){
	//clock_t begin, end;

	clock_t begin = clock();
	for (int row = 0; row < MATRIX_SIZE; row++) {  
		for (int col = 0; col < MATRIX_SIZE; col++) {  
			for (int inner = 0; inner < MATRIX_SIZE; inner++) {  
				C[row][col] += A[row][inner] * B[inner][col];  
			}   
		}  
	} 
	clock_t end = clock();
	printf("CPU time: %llf sec\n", (double)(end - begin)/CLOCKS_PER_SEC);
}